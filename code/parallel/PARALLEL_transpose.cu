
#include <hip/hip_runtime.h>
#include <stdio.h>
//#include "gputimer.h"

const dim3 BLOCK_SIZE(32,32);

// For testing
unsigned int compare_matrices(unsigned int *gpu, unsigned int *ref,
                              const unsigned int ROWS, const unsigned int COLUMNS){
  unsigned int result = 0;

  for(unsigned int i=0; i < COLUMNS; i++)
    for(unsigned int j=0; j < ROWS; j++)
      if (ref[i + j*COLUMNS] != gpu[i + j*COLUMNS]){
        printf("reference(%d,%d) = %f but test(%d,%d) = %f\n",
               i,j,ref[i+j*COLUMNS],i,j,gpu[i+j*COLUMNS]);
        result = 1;
      }
  return result;
}

void fill_matrix(unsigned int * mat,
	             const unsigned int ROWS, const unsigned int COLUMNS){
	for(unsigned int i=0; i < ROWS * COLUMNS; i++)
		mat[i] = (unsigned int) i;
}

/* CPU KERNEL */
void transpose_CPU(unsigned int * in, unsigned int * out,
                   const unsigned int ROWS, const unsigned int COLUMNS){
	for(unsigned int row=0; row < ROWS; row++)
    	for(unsigned int column=0; column < COLUMNS; column++)
      		out[column + row*COLUMNS] = in[row + column*ROWS]; // out(j,i) = in(i,j)
}

/* KERNEL */
__global__
void transpose_kernel(unsigned int * d_out, unsigned int * d_in,
                      const unsigned int ROWS, const unsigned int COLUMNS){
  unsigned int row = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int column = threadIdx.y + blockIdx.y * blockDim.y;
  if((row >= ROWS) || (column >= COLUMNS)) return;

  d_out[column + row*COLUMNS] = d_in[row + column*ROWS];
}

int main(int argc, char **argv){
  const unsigned int ROWS = 1<<4,
                     COLUMNS = 1<<4,
                     BYTES_ARRAY = ROWS*COLUMNS*sizeof(unsigned int);

  unsigned int * h_in = (unsigned int *) malloc(BYTES_ARRAY),
               * h_out = (unsigned int *) malloc(BYTES_ARRAY),
               * gold = (unsigned int *) malloc(BYTES_ARRAY);

  fill_matrix(h_in, ROWS, COLUMNS);
  transpose_CPU(h_in, gold, ROWS, COLUMNS);

  unsigned int * d_in, * d_out;

  hipMalloc(&d_in, BYTES_ARRAY);
  hipMalloc(&d_out, BYTES_ARRAY);
  hipMemcpy(d_in, h_in, BYTES_ARRAY, hipMemcpyHostToDevice);

//  GpuTimer timer;

  /* STARTING KERNEL */

  const dim3 GRID_SIZE(ROWS/BLOCK_SIZE.x + 1, COLUMNS/BLOCK_SIZE.y + 1);
//  timer.start();
  transpose_kernel<<<GRID_SIZE, BLOCK_SIZE>>>(d_out, d_in, ROWS, COLUMNS);
//  timer.stop();

  hipMemcpy(h_out, d_out, BYTES_ARRAY, hipMemcpyDeviceToHost);
  printf("transpose_serial\nVerifying transpose...%s\n", 
           compare_matrices(h_out, gold, ROWS, COLUMNS) ? "Failed" : "Success");

  hipFree(d_in);
  hipFree(d_out);
}